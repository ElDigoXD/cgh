#include "hip/hip_runtime.h"
#include "test.h"

#include <array>
#include <iostream>
#include <vector>
#include <cuda/std/complex>

#include "Color.h"
#include "Vector.h"


#define IMAGE_WIDTH 1920
#define IMAGE_HEIGHT 1080
static constexpr uint num_pixels = IMAGE_WIDTH * IMAGE_HEIGHT;
#define CU(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << result << " at " <<
                file << ":" << line << " '" << func << "' \n";

        std::cerr << hipGetErrorName(result) << std::endl;
        std::cerr << hipGetErrorString(result) << std::endl;

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

template<typename REAL_T>
__device__ REAL_T distance(const Point &a, const Point &b) {
    return std::sqrt(
        (static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x)) * (static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x)) +
        (static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y)) * (static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y)) +
        (static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z)) * (static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z))
    );
}

__global__ void a(cuda::std::complex<double> *complex_pixels, unsigned char *pixels, const std::vector<std::tuple<Point, Color, float> > &point_cloud, const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    const uint x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= IMAGE_WIDTH) || (y >= IMAGE_HEIGHT)) return;
    const uint pixel_index = y * IMAGE_WIDTH + x;

    typedef float REAL_T;
    typedef cuda::std::complex<REAL_T> COMPLEX_T;
    static constexpr REAL_T two_pi_over_wavelength = 2 * std::numbers::pi / 0.0006328;

    const auto slm_pixel_center = slm_pixel_00_location + (slm_pixel_delta_x * x) + (slm_pixel_delta_y * y);
    COMPLEX_T agg;
    for (const auto &[point, color, phase]: point_cloud) {
        const auto amplitude = luminance(color);
        const auto sub_phase = two_pi_over_wavelength * distance<REAL_T>(slm_pixel_center, point) + phase;
        const auto sub_phase_c = cuda::std::polar(static_cast<REAL_T>(1), sub_phase);
        const auto wave = amplitude * sub_phase_c;
        agg += wave;
    }

    complex_pixels[pixel_index] = agg / static_cast<REAL_T>(point_cloud.size());
    const auto a = static_cast<unsigned char>((arg(agg) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    pixels[pixel_index * 4 + 0] = a;
    pixels[pixel_index * 4 + 1] = a;
    pixels[pixel_index * 4 + 2] = a;
    pixels[pixel_index * 4 + 3] = 255;
}

__host__ void use_cuda(unsigned char pixels[], std::complex<Real> complex_pixels[], const std::vector<std::tuple<Point, Color, float> > &point_cloud, Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    dim3 block(32, 32);
    dim3 grid(IMAGE_WIDTH / block.x + 1, IMAGE_HEIGHT / block.y + 1);

    unsigned char *pixels_buff;
    cuda::std::complex<double> *complex_pixels_buff;
    CU(hipMallocManaged(&complex_pixels_buff, num_pixels * sizeof(cuda::std::complex<double>)));
    CU(hipMallocManaged(&pixels_buff, num_pixels * 4 * sizeof(unsigned char)));

    a<<<grid, block>>>(complex_pixels_buff, pixels_buff, point_cloud, slm_pixel_00_location, slm_pixel_delta_x, slm_pixel_delta_y);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());
    std::copy_n(pixels_buff, num_pixels * 4, pixels);
    std::copy_n(complex_pixels_buff, num_pixels, complex_pixels);
    CU(hipFree(pixels_buff));
    CU(hipFree(complex_pixels_buff));
}
