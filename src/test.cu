#include "hip/hip_runtime.h"
#include "test.h"

#include <array>
#include <iostream>
#include <vector>
#include <cuda/std/complex>

#include "Color.h"
#include "Vecf.h"
#include "Vector.h"


#define IMAGE_WIDTH 1920
#define IMAGE_HEIGHT 1080
static constexpr uint num_pixels = IMAGE_WIDTH * IMAGE_HEIGHT;
#define CU(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(const hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << result << " at " <<
                file << ":" << line << " '" << func << "' \n";

        std::cerr << hipGetErrorName(result) << std::endl;
        std::cerr << hipGetErrorString(result) << std::endl;

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

template<typename REAL_T>
__device__ REAL_T distance(const Point &a, const Point &b) {
    return std::sqrt(
        (static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x)) * (static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x)) +
        (static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y)) * (static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y)) +
        (static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z)) * (static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z))
    );
}

__global__ void kernel(cuda::std::complex<double> *complex_pixels, unsigned char *pixels, const std::vector<std::tuple<Point, Color, float> > &point_cloud, const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    const uint x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= IMAGE_WIDTH) || (y >= IMAGE_HEIGHT)) return;
    const uint pixel_index = y * IMAGE_WIDTH + x;

    typedef float REAL_T;
    typedef cuda::std::complex<REAL_T> COMPLEX_T;
    static constexpr REAL_T two_pi_over_wavelength_red = 2 * std::numbers::pi / 0.0006328; // Helium–neon laser
    static constexpr REAL_T two_pi_over_wavelength_green = 2 * std::numbers::pi / 0.000532; // Nd:YAG laser
    static constexpr REAL_T two_pi_over_wavelength_blue = 2 * std::numbers::pi / 0.000441563; // Helium–cadmium laser

    const auto slm_pixel_center = slm_pixel_00_location + (slm_pixel_delta_x * x) + (slm_pixel_delta_y * y);
    COMPLEX_T agg_luminance = 0;
    COMPLEX_T agg_red;
    COMPLEX_T agg_green;
    COMPLEX_T agg_blue;
    for (const auto &[point, color, phase]: point_cloud) {
        const Vecf color2 = Vecf{color.data};
        const auto amplitude = static_cast<REAL_T>(luminance(color));
        const auto sub_phase = two_pi_over_wavelength_red * distance<REAL_T>(slm_pixel_center, point) + phase;
        const auto sub_phase_c = cuda::std::polar(static_cast<REAL_T>(1), sub_phase);
        const auto wave = amplitude * sub_phase_c;
        agg_luminance += wave;
        const auto distance_to_point = distance<REAL_T>(slm_pixel_center, point);
        const auto sub_phase_red = two_pi_over_wavelength_red * distance_to_point + phase;
        const auto sub_phase_c_red = cuda::std::polar(static_cast<REAL_T>(1), sub_phase_red);
        const auto wave_red = color2.r * sub_phase_c_red;
        agg_red += wave_red;
        const auto sub_phase_green = two_pi_over_wavelength_green * distance_to_point + phase;
        const auto sub_phase_c_green = cuda::std::polar(static_cast<REAL_T>(1), sub_phase_green);
        const auto wave_green = color2.g * sub_phase_c_green;
        agg_green += wave_green;
        const auto sub_phase_blue = two_pi_over_wavelength_blue * distance_to_point + phase;
        const auto sub_phase_c_blue = cuda::std::polar(static_cast<REAL_T>(1), sub_phase_blue);
        const auto wave_blue = color2.b * sub_phase_c_blue;
        agg_blue += wave_blue;
    }

    // todo: luminance in A
    complex_pixels[pixel_index] = agg_luminance / static_cast<REAL_T>(point_cloud.size());
    // auto a = static_cast<unsigned char>((arg(agg_luminance) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    // pixels[pixel_index * 4 + 0] = a;
    // pixels[pixel_index * 4 + 1] = a;
    // pixels[pixel_index * 4 + 2] = a;
    pixels[pixel_index * 4 + 0] = static_cast<unsigned char>((arg(agg_red) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    pixels[pixel_index * 4 + 1] = static_cast<unsigned char>((arg(agg_green) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    pixels[pixel_index * 4 + 2] = static_cast<unsigned char>((arg(agg_blue) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    pixels[pixel_index * 4 + 3] = 255;
}

__host__ void use_cuda(unsigned char pixels[], std::complex<Real> complex_pixels[], const std::vector<std::tuple<Point, Color, float> > &point_cloud, Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    dim3 block(32, 32);
    dim3 grid(IMAGE_WIDTH / block.x + 1, IMAGE_HEIGHT / block.y + 1);

    unsigned char *pixels_buff;
    cuda::std::complex<double> *complex_pixels_buff;
    CU(hipMallocManaged(&complex_pixels_buff, num_pixels * sizeof(cuda::std::complex<double>)));
    CU(hipMallocManaged(&pixels_buff, num_pixels * 4 * sizeof(unsigned char)));

    kernel<<<grid, block>>>(complex_pixels_buff, pixels_buff, point_cloud, slm_pixel_00_location, slm_pixel_delta_x, slm_pixel_delta_y);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());
    std::copy_n(pixels_buff, num_pixels * 4, pixels);
    std::copy_n(complex_pixels_buff, num_pixels, complex_pixels);
    CU(hipFree(pixels_buff));
    CU(hipFree(complex_pixels_buff));
}
