#include "hip/hip_runtime.h"
#include "config.h"

#include <algorithm>
#include <array>
#include <iostream>
#include <vector>
#include <cuda/std/complex>


#include "PointCloud.h"
#include "Vecf.h"
#include "Vector.h"

static constexpr uint num_pixels = IMAGE_WIDTH * IMAGE_HEIGHT;
#define CU(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(const hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << result << " at " <<
                file << ":" << line << " '" << func << "' \n";

        std::cerr << hipGetErrorName(result) << std::endl;
        std::cerr << hipGetErrorString(result) << std::endl;

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

template<typename REAL_T>
__device__ REAL_T distance(const Point &a, const Point &b) {
    return std::sqrt(
        (static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x)) * (static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x)) +
        (static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y)) * (static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y)) +
        (static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z)) * (static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z))
    );
}

template<typename REAL_T>
__device__ cuda::std::complex<REAL_T> compute_wave(REAL_T two_pi_over_wavelength, REAL_T distance_to_point, REAL_T amplitude, REAL_T phase) {
    const auto sub_phase_red = two_pi_over_wavelength * distance_to_point + phase;
    const auto sub_phase_c = cuda::std::polar(static_cast<REAL_T>(1), sub_phase_red);
    return amplitude * sub_phase_c;
}

__global__ void kernel(cuda::std::complex<double> *complex_pixels, unsigned char *pixels, const PointCloud &point_cloud, const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x,
                       const Vec &slm_pixel_delta_y) {
    const uint x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= IMAGE_WIDTH) || (y >= IMAGE_HEIGHT)) return;
    const uint pixel_index = y * IMAGE_WIDTH + x;

    typedef double REAL_T;
    typedef cuda::std::complex<REAL_T> COMPLEX_T;
    static constexpr REAL_T two_pi_over_wavelength_red = 2 * std::numbers::pi / 0.0006328; // Helium–neon laser
    static constexpr REAL_T two_pi_over_wavelength_green = 2 * std::numbers::pi / 0.000532; // Nd:YAG laser
    static constexpr REAL_T two_pi_over_wavelength_blue = 2 * std::numbers::pi / 0.000441563; // Helium–cadmium laser

    const auto slm_pixel_center = slm_pixel_00_location + (slm_pixel_delta_x * x) + (slm_pixel_delta_y * y);
    COMPLEX_T agg_luminance = 0;
    COMPLEX_T agg_red;
    COMPLEX_T agg_green;
    COMPLEX_T agg_blue;
    for (const auto &[point, color, phase]: point_cloud) {
        const auto distance_to_point = distance<REAL_T>(slm_pixel_center, point);
        agg_luminance += compute_wave<REAL_T>(two_pi_over_wavelength_red, distance_to_point, luminance(color), phase);
        // agg_red += compute_wave<REAL_T>(two_pi_over_wavelength_red, distance_to_point, color.r, phase);
        // agg_green += compute_wave<REAL_T>(two_pi_over_wavelength_green, distance_to_point, color.g, phase);
        // agg_blue += compute_wave<REAL_T>(two_pi_over_wavelength_blue, distance_to_point, color.b, phase);
    }

    // todo: luminance in A
    complex_pixels[pixel_index] = agg_luminance / static_cast<REAL_T>(point_cloud.size());
    // auto a = static_cast<unsigned char>((arg(agg_luminance) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    // pixels[pixel_index * 4 + 0] = a;
    // pixels[pixel_index * 4 + 1] = a;
    // pixels[pixel_index * 4 + 2] = a;
    // pixels[pixel_index * 4 + 0] = static_cast<unsigned char>((arg(agg_red) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    // pixels[pixel_index * 4 + 1] = static_cast<unsigned char>((arg(agg_green) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    // pixels[pixel_index * 4 + 2] = static_cast<unsigned char>((arg(agg_blue) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    // pixels[pixel_index * 4 + 3] = 255;
}

__host__ void use_cuda(unsigned char pixels[], std::complex<Real> complex_pixels[], const PointCloud &point_cloud, const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    dim3 block(32, 32);
    dim3 grid(IMAGE_WIDTH / block.x + 1, IMAGE_HEIGHT / block.y + 1);

    unsigned char *pixels_buff;
    cuda::std::complex<double> *complex_pixels_buff;
    CU(hipMallocManaged(&complex_pixels_buff, num_pixels * sizeof(cuda::std::complex<double>)));
    //CU(hipMallocManaged(&pixels_buff, num_pixels * 4 * sizeof(unsigned char)));

    kernel<<<grid, block>>>(complex_pixels_buff, /* pixels_buff */ nullptr, point_cloud, slm_pixel_00_location, slm_pixel_delta_x, slm_pixel_delta_y);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());
    //std::copy_n(pixels_buff, num_pixels * 4, pixels);
    std::copy_n(complex_pixels_buff, num_pixels, complex_pixels);
    //CU(hipFree(pixels_buff));
    CU(hipFree(complex_pixels_buff));
}
