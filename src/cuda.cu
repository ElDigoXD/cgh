#include "hip/hip_runtime.h"
#include "config.h"

#include <algorithm>
#include <array>
#include <iostream>
#include <vector>
#include <cuda/std/complex>


#include "PointCloud.h"
#include "Vecf.h"
#include "Vector.h"

#define CU(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(const hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << result << " at " <<
                file << ":" << line << " '" << func << "' \n";

        std::cerr << hipGetErrorName(result) << std::endl;
        std::cerr << hipGetErrorString(result) << std::endl;

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

template<typename REAL_T>
__device__ REAL_T inline distance(const Point &a, const Point &b) {
    const REAL_T dx = static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x);
    const REAL_T dy = static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y);
    const REAL_T dz = static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z);

    if constexpr (std::is_same_v<REAL_T, float>) {
        return norm3df(dx, dy, dz);
    } else {
        return norm3d(dx, dy, dz);
    }
}

template<typename REAL_T>
__device__ inline cuda::std::complex<REAL_T> compute_wave(REAL_T two_pi_over_wavelength, REAL_T distance_to_point, REAL_T amplitude, float phase) {
    const auto sub_phase_red = two_pi_over_wavelength * distance_to_point + phase;
    REAL_T sin_val, cos_val;
    if constexpr (std::is_same_v<REAL_T, float>) {
        sincosf(sub_phase_red, &sin_val, &cos_val);
    } else {
        sincos(sub_phase_red, &sin_val, &cos_val);
    }
    return amplitude * cuda::std::complex<REAL_T>{ cos_val, sin_val};
}

typedef float REAL_T;
typedef cuda::std::complex<REAL_T> COMPLEX_T;

__constant__ float SCALE = 255.f / (2 * M_PIf);
__constant__ REAL_T two_pi_over_wavelength_red = 2 * M_PI / 0.0006328; // Helium–neon laser
__constant__ REAL_T two_pi_over_wavelength_green = 2 * M_PI / 0.000532; // Nd:YAG laser
__constant__ REAL_T two_pi_over_wavelength_blue = 2 * M_PI / 0.000441563; // Helium–cadmium laser

__global__ void kernel(cuda::std::complex<double> *complex_pixels, unsigned char *pixels, const PointCloud &point_cloud, const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x,
                       const Vec &slm_pixel_delta_y) {
    const uint x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= IMAGE_WIDTH) || (y >= IMAGE_HEIGHT)) return;
    const uint pixel_index = y * IMAGE_WIDTH + x;


    const auto slm_pixel_center = slm_pixel_00_location + (slm_pixel_delta_x * x) + (slm_pixel_delta_y * y);

    COMPLEX_T agg_luminance, agg_red, agg_green, agg_blue;
    for (const auto &[point, color, phase]: point_cloud) {
        const auto distance_to_point = distance<REAL_T>(slm_pixel_center, point);
        agg_luminance += compute_wave<REAL_T>(two_pi_over_wavelength_red, distance_to_point, luminance(color), phase);
#if ENABLE_COLOR_CGH
        agg_red += compute_wave<REAL_T>(two_pi_over_wavelength_red, distance_to_point, color.r, phase);
        agg_green += compute_wave<REAL_T>(two_pi_over_wavelength_green, distance_to_point, color.g, phase);
        agg_blue += compute_wave<REAL_T>(two_pi_over_wavelength_blue, distance_to_point, color.b, phase);
#endif // #if ENABLE_COLOR_CGH
    }

#if ENABLE_COLOR_CGH
    pixels[pixel_index * 4 + 0] = static_cast<unsigned char>((arg(agg_red) + M_PIf) * SCALE);
    pixels[pixel_index * 4 + 1] = static_cast<unsigned char>((arg(agg_green) + M_PIf) * SCALE);
    pixels[pixel_index * 4 + 2] = static_cast<unsigned char>((arg(agg_blue) + M_PIf) * SCALE);
    pixels[pixel_index * 4 + 3] = static_cast<unsigned char>((arg(agg_luminance) + M_PIf) * SCALE);
#else // #if ENABLE_COLOR_CGH
    const auto luminance = agg_luminance / static_cast<REAL_T>(point_cloud.size());
    complex_pixels[pixel_index] = luminance;
#if VIRTUAL_SLM_FACTOR == 1
    const auto a = static_cast<uint8_t>((arg(luminance) + std::numbers::pi) / (2 * std::numbers::pi) * 255);
    pixels[pixel_index * 4 + 0] = a;
    pixels[pixel_index * 4 + 1] = a;
    pixels[pixel_index * 4 + 2] = a;
    pixels[pixel_index * 4 + 3] = 255;
#endif // #if VIRTUAL_SLM_FACTOR == 1
#endif // #if ENABLE_COLOR_CGH #else
}

__host__ void use_cuda(unsigned char pixels[], std::complex<Real> complex_pixels[], const PointCloud &point_cloud, const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    static constexpr uint num_pixels = IMAGE_WIDTH * IMAGE_HEIGHT;

    dim3 block(32, 32);
    dim3 grid(IMAGE_WIDTH / block.x + 1, IMAGE_HEIGHT / block.y + 1);

    unsigned char *pixels_buff;
    cuda::std::complex<double> *complex_pixels_buff;
#if ENABLE_COLOR_CGH
    CU(hipMallocManaged(&complex_pixels_buff, num_pixels * sizeof(cuda::std::complex<double>)));
#endif // #if ENABLE_COLOR_CGH
    CU(hipMallocManaged(&pixels_buff, num_pixels * 4 * sizeof(unsigned char)));
#if VIRTUAL_SLM_FACTOR == 1
    CU(hipMallocManaged(&pixels_buff, num_pixels * 4 * sizeof(unsigned char)));
#endif // #if VIRTUAL_SLM_FACTOR == 1
    kernel<<<grid, block>>>(complex_pixels_buff, pixels_buff, point_cloud, slm_pixel_00_location, slm_pixel_delta_x, slm_pixel_delta_y);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());
    std::copy_n(pixels_buff, num_pixels * 4, pixels);
    std::copy_n(complex_pixels_buff, num_pixels, complex_pixels);
    CU(hipFree(pixels_buff));
    CU(hipFree(complex_pixels_buff));
}
