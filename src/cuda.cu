#include "hip/hip_runtime.h"
#include "config.h"

#include <algorithm>
#include <array>
#include <iostream>
#include <vector>
#include <cuda/std/complex>


#include "PointCloud.h"
#include "utils.h"
#include "Vecf.h"
#include "Vector.h"

#define CU(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(const hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << result << " at " <<
                file << ":" << line << " '" << func << "' \n";

        std::cerr << hipGetErrorName(result) << std::endl;
        std::cerr << hipGetErrorString(result) << std::endl;

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

template<typename REAL_T>
__device__ REAL_T distance(const Point &a, const Point &b) {
    const REAL_T dx = static_cast<REAL_T>(a.x) - static_cast<REAL_T>(b.x);
    const REAL_T dy = static_cast<REAL_T>(a.y) - static_cast<REAL_T>(b.y);
    const REAL_T dz = static_cast<REAL_T>(a.z) - static_cast<REAL_T>(b.z);

    if constexpr (std::is_same_v<REAL_T, float>) {
        return norm3df(dx, dy, dz);
    } else {
        return norm3d(dx, dy, dz);
    }
}

template<typename REAL_T>
__device__ cuda::std::complex<REAL_T> compute_wave(const REAL_T one_over_wavelength, const REAL_T distance_to_point,
                                                   const REAL_T amplitude, const float phase) {
    REAL_T sin_val, cos_val;
    if constexpr (std::is_same_v<REAL_T, float>) {
        const double sub_phase = 1.0 * one_over_wavelength * distance_to_point;
        const float y = fmaf(sub_phase - floor(sub_phase), 2.f, phase);
        sincospif(y, &sin_val, &cos_val);
    } else {
        const double y = one_over_wavelength * distance_to_point + phase;
        sincospi(y, &sin_val, &cos_val);
    }
    return {cos_val * amplitude, sin_val * amplitude};
}

using REAL_T = float;
using COMPLEX_T = cuda::std::complex<REAL_T>;

__constant__ float SCALE = 255.f / (2.f * M_PIf);
__constant__ REAL_T one_over_wavelength_red = 1 / 0.0006328f; // Helium–neon laser
__constant__ REAL_T one_over_wavelength_green = 1 / 0.000532f; // Nd:YAG laser
__constant__ REAL_T one_over_wavelength_blue = 1 / 0.000441563f; // Helium–cadmium laser

__global__ void kernel(cuda::std::complex<double> *out_complex_pixels, unsigned char *out_pixels,
                       const PointCloudPoint *point_cloud, const unsigned int pc_size,
                       const Point slm_pixel_00_location, const Vec slm_pixel_delta_x, const Vec slm_pixel_delta_y) {
    const uint x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= IMAGE_WIDTH) || (y >= IMAGE_HEIGHT)) return;
    const uint pixel_index = y * IMAGE_WIDTH + x;


    const auto slm_pixel_center = slm_pixel_00_location + (slm_pixel_delta_x * x) + (slm_pixel_delta_y * y);

    COMPLEX_T agg_luminance, agg_red, agg_green, agg_blue;
    for (unsigned int i = 0; i < pc_size; i++) {
        const auto [point, color, phase] = point_cloud[i];
        const auto distance_to_point = distance<REAL_T>(slm_pixel_center, point);
        agg_luminance += compute_wave<REAL_T>(one_over_wavelength_red, distance_to_point, luminance(color), phase);
#if ENABLE_COLOR_CGH
        agg_red += compute_wave<REAL_T>(one_over_wavelength_red, distance_to_point, color.r, phase);
        agg_green += compute_wave<REAL_T>(one_over_wavelength_green, distance_to_point, color.g, phase);
        agg_blue += compute_wave<REAL_T>(one_over_wavelength_blue, distance_to_point, color.b, phase);
#endif // #if ENABLE_COLOR_CGH
    }

#if ENABLE_COLOR_CGH
    out_pixels[pixel_index * 4 + 0] = static_cast<unsigned char>((arg(agg_red) + M_PIf) * SCALE);
    out_pixels[pixel_index * 4 + 1] = static_cast<unsigned char>((arg(agg_green) + M_PIf) * SCALE);
    out_pixels[pixel_index * 4 + 2] = static_cast<unsigned char>((arg(agg_blue) + M_PIf) * SCALE);
    out_pixels[pixel_index * 4 + 3] = static_cast<unsigned char>((arg(agg_luminance) + M_PIf) * SCALE);
    //out_pixels[pixel_index * 4 + 3] = 255;
#else // #if ENABLE_COLOR_CGH
#if VIRTUAL_SLM_FACTOR == 1
    const auto l = static_cast<unsigned char>((arg(agg_luminance) + M_PIf) * SCALE);
    out_pixels[pixel_index * 4 + 0] = l;
    out_pixels[pixel_index * 4 + 1] = l;
    out_pixels[pixel_index * 4 + 2] = l;
    out_pixels[pixel_index * 4 + 3] = 255;
#elif VIRTUAL_SLM_FACTOR > 1
    const auto luminance = agg_luminance / static_cast<REAL_T>(point_cloud.size());
    const auto a =  static_cast<unsigned char>((arg(luminance) + M_PIf) * SCALE);
    out_pixels[pixel_index * 4 + 0] = a;
    out_pixels[pixel_index * 4 + 1] = a;
    out_pixels[pixel_index * 4 + 2] = a;
    out_pixels[pixel_index * 4 + 3] = 255;
#endif // #if VIRTUAL_SLM_FACTOR
#endif // #if ENABLE_COLOR_CGH #else
}

// Point cloud phase must be in the range [0, 2).
__host__ void use_cuda(unsigned char out_pixels[], std::complex<Real> out_complex_pixels[], const PointCloud &point_cloud,
                       const Point &slm_pixel_00_location, const Vec &slm_pixel_delta_x, const Vec &slm_pixel_delta_y) {
    static constexpr uint num_pixels = IMAGE_WIDTH * IMAGE_HEIGHT;


    //const float x = 1.0 * (1 / 0.0006328f) * 310.0f;
    //const float y = x - floor(x);
    //printf ("smallest representable difference near %.8f is %.16f | required: %f\n", x, x - nextafter(x, 0.0f), 1/256.0f);
    // printf ("smallest representable difference near %.8f is %.16f | required: %f\n", y, y - nextafter(y, 0.0f), 1/256.0f);
    //return;


    printf("         Using: CUDA\n");
    printf("         Precision: %s\n", sizeof(REAL_T) == sizeof(float) ? "single" : "double");
    printf("         Image size: %s x %s (factor %d)\n", add_thousand_separator(IMAGE_WIDTH).c_str(),
           add_thousand_separator(IMAGE_HEIGHT).c_str(), VIRTUAL_SLM_FACTOR);
    printf("         Num points: %s\n", add_thousand_separator(point_cloud.size()).c_str());
    printf("         Enable color: %s\n", ENABLE_COLOR_CGH ? "true" : "false");
    dim3 block(32, 32);
    dim3 grid(IMAGE_WIDTH / block.x + 1, IMAGE_HEIGHT / block.y + 1);

    unsigned char *out_pixels_buff;
    cuda::std::complex<double> *complex_pixels_buff;

#if VIRTUAL_SLM_FACTOR > 1
    CU(hipMallocManaged(&complex_pixels_buff, num_pixels * sizeof(cuda::std::complex<double>)));
#endif
    CU(hipMallocManaged(&out_pixels_buff, num_pixels * 4 * sizeof(unsigned char)));
    PointCloudPoint *pc;
    CU(hipMallocManaged(&pc, point_cloud.size() * sizeof(PointCloudPoint)));
    for (unsigned int i = 0; i < point_cloud.size(); i++) {
        pc[i] = point_cloud[i];
    }
    CU(hipGetLastError());
#if VIRTUAL_SLM_FACTOR == 1
    //CU(hipMallocManaged(&out_pixels_buff, num_pixels * 4 * sizeof(unsigned char)));
#endif // #if VIRTUAL_SLM_FACTOR == 1
    kernel<<<grid, block>>>(complex_pixels_buff, out_pixels_buff, pc, point_cloud.size(), slm_pixel_00_location, slm_pixel_delta_x,
                            slm_pixel_delta_y);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());
    std::copy_n(out_pixels_buff, num_pixels * 4, out_pixels);
    CU(hipFree(out_pixels_buff));
#if VIRTUAL_SLM_FACTOR > 1
    std::copy_n(complex_pixels_buff, num_pixels, out_complex_pixels);
    CU(hipFree(complex_pixels_buff));
#endif
}
